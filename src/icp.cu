#include "hip/hip_runtime.h"
#include "icp.hpp"
#include "svd.hpp"
#include <thrust/gather.h>
#include <thrust/inner_product.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>

void ICP::setTarget(std::vector<float3> const &target, hipStream_t stream) {
    kdtree.buildTree(target, stream);
}

struct SubtractFunctor {
    __host__ __device__ SubtractFunctor(float value) : value(value) {}
    inline __host__ __device__ float operator()(float x) const { return x - value; }

  private:
    float value;
};

// in-place operation
__global__ void applyTransformation(float3 *source, uint32_t n_source, float *R, float *t) {
    int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n_source)
        return;
    // R: 3x3 matrix
    // t: 3x1 vector
    float3 s = source[idx];
    float3 r;
    r.x = R[0] * s.x + R[1] * s.y + R[2] * s.z + t[0];
    r.y = R[3] * s.x + R[4] * s.y + R[5] * s.z + t[1];
    r.z = R[6] * s.x + R[7] * s.y + R[8] * s.z + t[2];
    source[idx] = r;
}

// in-place operation, this is for estimating error
__global__ void applyTransformation2(float *dsx, float *dsy, float *dsz, float *dtx, float *dty, float *dtz,
                                     uint32_t start, uint32_t end, float *R, float *t) {
    int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < start || idx >= end)
        return;
    // Transform the source points (dsx, dsy, dsz)
    // R: 3x3 matrix
    // t: 3x1 vector
    float sx = R[0] * dsx[idx] + R[1] * dsy[idx] + R[2] * dsz[idx] + t[0];
    float sy = R[3] * dsx[idx] + R[4] * dsy[idx] + R[5] * dsz[idx] + t[1];
    float sz = R[6] * dsx[idx] + R[7] * dsy[idx] + R[8] * dsz[idx] + t[2];
    dsx[idx] = (sx - dtx[idx]) * (sx - dtx[idx]);
    dsy[idx] = (sy - dty[idx]) * (sy - dty[idx]);
    dsz[idx] = (sz - dtz[idx]) * (sz - dtz[idx]);
}

// Get the next transformation
// gR, R: 3x3 matrix (global and current rotation)
// gt, t: 3x1 vector (global and current translation)
std::tuple<std::vector<float>, std::vector<float>> getTranformation(std::vector<float> const &gR,
                                                                    std::vector<float> const &gt,
                                                                    std::vector<float> const &R,
                                                                    std::vector<float> const &t) {
    // Next transformation
    std::vector<float> nR(9, 0); // R * gR
    std::vector<float> nt(3, 0); // R * gt + t

    for (int i = 0; i < 3; ++i) {
        for (int j = 0; j < 3; ++j)
            nR[3 * i + j] += R[3 * i] * gR[j] + R[3 * i + 1] * gR[j + 3] + R[3 * i + 2] * gR[j + 6];
        nt[i] += R[3 * i] * gt[0] + R[3 * i + 1] * gt[1] + R[3 * i + 2] * gt[2] + t[i];
    }
    return {nR, nt};
}

// Frobenius norm: ||A||_F = sqrt(Trace(A * A^T))
float getTranformationError(std::vector<float> const &gR, std::vector<float> const &gt,
                            std::vector<float> const &nR, std::vector<float> const &nt) {
    float error = 0.0f;
    for (int i = 0; i < 9; ++i)
        error += (nR[i] - gR[i]) * (nR[i] - gR[i]);
    for (int i = 0; i < 3; ++i)
        error += (nt[i] - gt[i]) * (nt[i] - gt[i]);
    return sqrt(error);
}

// Reference: https://learnopencv.com/iterative-closest-point-icp-explained/
std::tuple<bool, float> ICP::align(std::vector<float3> const &source, float maxCorrespondenceDistance,
                                   int maximumIterations, float transformationEpsilon,
                                   float euclideanFitnessEpsilon, std::vector<float> &Rt,
                                   hipStream_t stream) {
    uint32_t n_source = source.size();
    thrust::device_vector<float3> d_source(source.begin(), source.end());
    thrust::device_vector<uint32_t> inlier(n_source, 0);

    // Allocate cuda memory for the source and target points
    thrust::device_vector<float> dsx(n_source), dsy(n_source), dsz(n_source);
    thrust::device_vector<float> dtx(n_source), dty(n_source), dtz(n_source);

    // For gathering the inliers
    thrust::device_vector<float> gsx(n_source), gsy(n_source), gsz(n_source);
    thrust::device_vector<float> gtx(n_source), gty(n_source), gtz(n_source);

    // R and t
    std::vector<float> gR(9, 0.0f);
    std::vector<float> gt(3, 0.0f);
    // Rt 4x4 matrix
    for (int i = 0; i < 3; ++i) {
        for (int j = 0; j < 3; ++j)
            gR[3 * i + j] = Rt[i * 4 + j];
        gt[i] = Rt[i * 4 + 3];
    }
    thrust::device_vector<float> dR(9);
    thrust::device_vector<float> dt(3);

    hipMemcpy(thrust::raw_pointer_cast(dR.data()), gR.data(), 9 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(thrust::raw_pointer_cast(dt.data()), gt.data(), 3 * sizeof(float), hipMemcpyHostToDevice);
    // Apply the initial transformation
    uint32_t blockSize = 1 << 8;
    uint32_t numBlocks = (n_source + blockSize - 1) / blockSize;
    applyTransformation<<<numBlocks, blockSize, 0, stream>>>(thrust::raw_pointer_cast(d_source.data()),
                                                             n_source, thrust::raw_pointer_cast(dR.data()),
                                                             thrust::raw_pointer_cast(dt.data()));
    hipStreamSynchronize(stream);

    float prevError = std::numeric_limits<float>::max();
    bool converged = false;
    float percentageInliers = 0.0f;

    auto policy = thrust::device.on(stream);
    for (int i = 0; i < maximumIterations; ++i) {
        // 1. Find correspondences
        kdtree.findCorrespondences(thrust::raw_pointer_cast(d_source.data()), n_source,
                                   maxCorrespondenceDistance, thrust::raw_pointer_cast(inlier.data()),
                                   thrust::raw_pointer_cast(dsx.data()), thrust::raw_pointer_cast(dsy.data()),
                                   thrust::raw_pointer_cast(dsz.data()), thrust::raw_pointer_cast(dtx.data()),
                                   thrust::raw_pointer_cast(dty.data()), thrust::raw_pointer_cast(dtz.data()),
                                   stream);
        hipStreamSynchronize(stream);

        // 2. Compute centroids
        // in-place scan: inlier[i] += inlier[i-1]
        thrust::inclusive_scan(policy, inlier.begin(), inlier.end(), inlier.begin());
        int32_t count; // number of inliers
        thrust::copy(inlier.end() - 1, inlier.end(), &count);
        if (count < 2)
            break;     // no inliers
        int32_t start; // start of inliers
        thrust::copy(inlier.begin(), inlier.begin() + 1, &start);
        start = 1 - start;

        // move all inliers to the front
        thrust::gather(policy, inlier.begin(), inlier.end(), dsx.begin(), gsx.begin());
        thrust::gather(policy, inlier.begin(), inlier.end(), dsy.begin(), gsy.begin());
        thrust::gather(policy, inlier.begin(), inlier.end(), dsz.begin(), gsz.begin());
        thrust::gather(policy, inlier.begin(), inlier.end(), dtx.begin(), gtx.begin());
        thrust::gather(policy, inlier.begin(), inlier.end(), dty.begin(), gty.begin());
        thrust::gather(policy, inlier.begin(), inlier.end(), dtz.begin(), gtz.begin());
        hipStreamSynchronize(stream);
        // compute centroids
        float csx = thrust::reduce(policy, gsx.begin() + start, gsx.begin() + count + start, 0.0f) / count;
        float csy = thrust::reduce(policy, gsy.begin() + start, gsy.begin() + count + start, 0.0f) / count;
        float csz = thrust::reduce(policy, gsz.begin() + start, gsz.begin() + count + start, 0.0f) / count;
        float ctx = thrust::reduce(policy, gtx.begin() + start, gtx.begin() + count + start, 0.0f) / count;
        float cty = thrust::reduce(policy, gty.begin() + start, gty.begin() + count + start, 0.0f) / count;
        float ctz = thrust::reduce(policy, gtz.begin() + start, gtz.begin() + count + start, 0.0f) / count;
        hipStreamSynchronize(stream);

        // 3. Center the points
        thrust::transform(policy, gsx.begin() + start, gsx.begin() + count + start, gsx.begin() + start,
                          SubtractFunctor(csx));
        thrust::transform(policy, gsy.begin() + start, gsy.begin() + count + start, gsy.begin() + start,
                          SubtractFunctor(csy));
        thrust::transform(policy, gsz.begin() + start, gsz.begin() + count + start, gsz.begin() + start,
                          SubtractFunctor(csz));
        thrust::transform(policy, gtx.begin() + start, gtx.begin() + count + start, gtx.begin() + start,
                          SubtractFunctor(ctx));
        thrust::transform(policy, gty.begin() + start, gty.begin() + count + start, gty.begin() + start,
                          SubtractFunctor(cty));
        thrust::transform(policy, gtz.begin() + start, gtz.begin() + count + start, gtz.begin() + start,
                          SubtractFunctor(ctz));
        hipStreamSynchronize(stream);

        // 4. Compute the covariance matrix
        std::vector<float> H(9);
        H[0] = thrust::inner_product(gsx.begin() + start, gsx.begin() + count + start, gtx.begin() + start,
                                     0.0f);
        H[1] = thrust::inner_product(gsx.begin() + start, gsx.begin() + count + start, gty.begin() + start,
                                     0.0f);
        H[2] = thrust::inner_product(gsx.begin() + start, gsx.begin() + count + start, gtz.begin() + start,
                                     0.0f);

        H[3] = thrust::inner_product(gsy.begin() + start, gsy.begin() + count + start, gtx.begin() + start,
                                     0.0f);
        H[4] = thrust::inner_product(gsy.begin() + start, gsy.begin() + count + start, gty.begin() + start,
                                     0.0f);
        H[5] = thrust::inner_product(gsy.begin() + start, gsy.begin() + count + start, gtz.begin() + start,
                                     0.0f);

        H[6] = thrust::inner_product(gsz.begin() + start, gsz.begin() + count + start, gtx.begin() + start,
                                     0.0f);
        H[7] = thrust::inner_product(gsz.begin() + start, gsz.begin() + count + start, gty.begin() + start,
                                     0.0f);
        H[8] = thrust::inner_product(gsz.begin() + start, gsz.begin() + count + start, gtz.begin() + start,
                                     0.0f);
        hipStreamSynchronize(stream);

        // 5. Compute Rotation and translation using SVD
        auto [R, t] = computeRt(H, csx, csy, csz, ctx, cty, ctz);
        hipMemcpy(thrust::raw_pointer_cast(dR.data()), R.data(), 9 * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(thrust::raw_pointer_cast(dt.data()), t.data(), 3 * sizeof(float), hipMemcpyHostToDevice);
        auto [nR, nt] = getTranformation(gR, gt, R, t);
        float error = getTranformationError(gR, gt, nR, nt);
        if (error < transformationEpsilon) {
            converged = true;
            percentageInliers = (float)count / n_source;
            break; // converged
        }
        gR = nR;
        gt = nt;

        // 6. Apply the transformation to the source points
        applyTransformation<<<numBlocks, blockSize, 0, stream>>>(
            thrust::raw_pointer_cast(d_source.data()), n_source, thrust::raw_pointer_cast(dR.data()),
            thrust::raw_pointer_cast(dt.data()));
        hipStreamSynchronize(stream);

        // 7. Compute the Euclidean distance error
        applyTransformation2<<<numBlocks, blockSize, 0, stream>>>(
            thrust::raw_pointer_cast(dsx.data()), thrust::raw_pointer_cast(dsy.data()),
            thrust::raw_pointer_cast(dsz.data()), thrust::raw_pointer_cast(dtx.data()),
            thrust::raw_pointer_cast(dty.data()), thrust::raw_pointer_cast(dtz.data()), start, count + start,
            thrust::raw_pointer_cast(dR.data()), thrust::raw_pointer_cast(dt.data()));
        hipStreamSynchronize(stream);
        float error2 = thrust::reduce(policy, dsx.begin() + start, dsx.begin() + count + start, 0.0f) +
                       thrust::reduce(policy, dsy.begin() + start, dsy.begin() + count + start, 0.0f) +
                       thrust::reduce(policy, dsz.begin() + start, dsz.begin() + count + start, 0.0f);
        hipStreamSynchronize(stream);
        error2 = sqrt(error2);
        if (abs(error2 - prevError) < euclideanFitnessEpsilon) {
            converged = true;
            percentageInliers = (float)count / n_source;
            break; // converged
        }
        prevError = error2;
    }
    // Copy the final transformation matrix to the output
    for (int i = 0; i < 3; ++i) {
        for (int j = 0; j < 3; ++j)
            Rt[i * 4 + j] = gR[3 * i + j];
        Rt[i * 4 + 3] = gt[i];
    }
    return {converged, percentageInliers};
}